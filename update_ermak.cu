#include "hip/hip_runtime.h"
#include "encabezados.h"

// ================================================== 2023-05-22 =====================
// HOST			(Ermak-McCammon)
// ===================================================================================


void update_ermak_hst(char type, float3 *rr_vec, float3 *rr_raw_vec,
                      float3 *ff_vec, gsl_rng *rand, parametros pars) {
    int ngrain;
    float3 drr_random, drr_total, ff, rr, rr_raw;
    float side, side_inv, dt, diameter, diameter_inverse;
    float temperature, amplitude;

    side = pars.side;
    side_inv = 1.0f / side;
    temperature = pars.temp_set;
    dt = pars.dt;

    if (type == 'b') {
        ngrain = pars.ngrain_big;
        diameter = pars.sigma_big;
        diameter_inverse = 1.0f / diameter;
    } else {
        ngrain = pars.ngrain_sml;
        diameter = pars.sigma_sml;
        diameter_inverse = 1.0f / diameter;
    }

    // amplitude of the random displacement
    amplitude = sqrt(2.0 * temperature * diameter_inverse * dt);

    for (int mm = 0; mm < ngrain; ++mm) {
        //fetch
        ff = ff_vec[mm];
        rr = rr_vec[mm];
        rr_raw = rr_raw_vec[mm];

        //random displacement

        drr_random.x = amplitude * gsl_ran_gaussian(rand, 1.0);
        drr_random.y = amplitude * gsl_ran_gaussian(rand, 1.0);
        drr_random.z = amplitude * gsl_ran_gaussian(rand, 1.0);

        // added force displacement

        drr_total.x = ff.x * diameter_inverse * dt + drr_random.x;
        drr_total.y = ff.y * diameter_inverse * dt + drr_random.y;
        drr_total.z = ff.z * diameter_inverse * dt + drr_random.z;

        //move position

        rr.x += drr_total.x;
        rr.y += drr_total.y;
        rr.z += drr_total.z;

        rr_raw.x += drr_total.x;
        rr_raw.y += drr_total.y;
        rr_raw.z += drr_total.z;

        //apply periodic boundary conditions

        rr.x -= side * floor(side_inv * rr.x);
        rr.y -= side * floor(side_inv * rr.y);
        rr.z -= side * floor(side_inv * rr.z);

        if (rr.x < 0.0001f && rr.y < 0.0001f && rr.z < 0.0001f)
            printf("Warning: Particle position at zero: (%f, %f, %f)\n", rr.x, rr.y, rr.z);
        //save

        rr_vec[mm] = rr;
        rr_raw_vec[mm] = rr_raw;

    }


}

// ===================================================================================
// DEVICE 		(Ermak-McCammon)
// ===================================================================================

#include <hiprand/hiprand_kernel.h>

__global__ void setup_rng_kernel(hiprandState *states, unsigned long seed, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N)
        hiprand_init(seed, idx, 0, &states[idx]);
}

__global__ void update_ermak_dev(char type, float3 *rr_vec, float3 *rr_raw_vec,
                      float3 *ff_vec, hiprandState *states, parametros pars) {
    int ngrain, mm;
    float3 drr_random, drr_total, ff, rr, rr_raw;
    float side, side_inv, dt, diameter, diameter_inverse;
    float temperature, amplitude;

    side = pars.side;
    side_inv = 1.0f / side;
    temperature = pars.temp_set;
    dt = pars.dt;

    if (type == 'b') {
        ngrain = pars.ngrain_big;
        diameter = pars.sigma_big;
        diameter_inverse = 1.0f / diameter;
    } else {
        ngrain = pars.ngrain_sml;
        diameter = pars.sigma_sml;
        diameter_inverse = 1.0f / diameter;
    }

    // amplitude of the random displacement
    amplitude = sqrt(2.0 * temperature * diameter_inverse * dt);

    mm = threadIdx.x + blockIdx.x * blockDim.x;

    if (mm < ngrain) {
        //fetch
        ff = ff_vec[mm];
        rr = rr_vec[mm];
        rr_raw = rr_raw_vec[mm];

        //random displacement

        drr_random.x = amplitude * hiprand_normal(&states[mm]);
        drr_random.y = amplitude * hiprand_normal(&states[mm]);
        drr_random.z = amplitude * hiprand_normal(&states[mm]);

        // added force displacement

        drr_total.x = ff.x * diameter_inverse * dt + drr_random.x;
        drr_total.y = ff.y * diameter_inverse * dt + drr_random.y;
        drr_total.z = ff.z * diameter_inverse * dt + drr_random.z;

        //move position

        rr.x += drr_total.x;
        rr.y += drr_total.y;
        rr.z += drr_total.z;

        rr_raw.x += drr_total.x;
        rr_raw.y += drr_total.y;
        rr_raw.z += drr_total.z;

        //apply periodic boundary conditions

        rr.x -= side * floor(side_inv * rr.x);
        rr.y -= side * floor(side_inv * rr.y);
        rr.z -= side * floor(side_inv * rr.z);

        //save

        rr_vec[mm] = rr;
        rr_raw_vec[mm] = rr_raw;

    }


}
